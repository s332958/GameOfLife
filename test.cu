#include "hip/hip_runtime.h"
#include "libs/mondo_kernel.cu"
#include "libs/NN_kernel.cu"
#include "libs/utils_kernel.cu"
#include "libs/utils_cpu.cpp"

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>

float random_float(float min, float max) {
    return min + static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * (max - min);
}

int calcola_weights(int *v, int n){
    int sum = 0;
    for(int i=0; i<n-1; i ++) {
        sum += (v[i]*v[i+1]);
    }
    return sum;
}

int calcola_biases(int *v, int n){
    int sum = 0;
    for(int i=1; i<n; i ++) {
        sum += (v[i]);
    }
    return sum;
}

int main() {

    srand(time(0));

    int const n          = 5;
    int world_dim        = 5;
    int numero_workspace = 10;
    int visione          = 3;
    int input_dim        = visione*visione*2;
    int output_dim       = (3*3)+1;
    int structure[n]     = {input_dim, 3, 2, 1, output_dim};
    int n_weights        = calcola_weights(structure,n);
    int n_baias          = calcola_biases(structure,n);
    int n_modelli        = 1;
    int n_step           = 10;

    hipStream_t stream = 0;

    size_t size_world_float               = world_dim*world_dim*sizeof(float);
    size_t size_world_int               = world_dim*world_dim*sizeof(int);
    size_t size_inputs              = numero_workspace*input_dim*sizeof(float);
    size_t size_outputs             = numero_workspace*output_dim*sizeof(float);
    size_t size_weight              = n_weights*n_modelli*sizeof(float);
    size_t size_bias                = n_baias*n_modelli*sizeof(float);
    size_t size_contribution_matrix = size_world_float*n_modelli*sizeof(float);

    float *world_value_h                = (float*)  malloc(size_world_float);
    float *world_signal_h               = (float*)  malloc(size_world_float);
    int   *world_id_h                   = (int*)    malloc(size_world_int);
    float *inputs_h                     = (float*)  malloc(size_inputs);
    float *outputs_h                    = (float*)  malloc(size_outputs);
    float *weight_h                     = (float*)  malloc(size_weight);
    float *baias_h                      = (float*)  malloc(size_bias);
    int   *alive_cell_h                 = (int*)    malloc(size_world_int);
    float *contribution_matrix_h        = (float*)  malloc(size_contribution_matrix);  
    int   alive_cell_count_h;

    float *world_value_d;   
    float *world_signal_d;   
    int   *world_id_d;         
    int   *alive_cell_d;       
    float *inputs_d;
    float *outputs_d;   
    float *weight_d;
    float *baias_d;     
    float *contribution_matrix_d;
    int   *alive_cell_count_d;
    // hipMallocManaged(&alive_cell_count_d, sizeof(int));

    hipMalloc((void**) &world_value_d, size_world_float);
    hipMalloc((void**) &world_signal_d, size_world_float);
    hipMalloc((void**) &world_id_d, size_world_int);
    hipMalloc((void**) &alive_cell_d, size_world_int);
    hipMalloc((void**) &inputs_d, size_inputs);
    hipMalloc((void**) &outputs_d, size_outputs);
    hipMalloc((void**) &weight_d, size_weight);
    hipMalloc((void**) &baias_d, size_bias);
    hipMalloc((void**) &contribution_matrix_d, size_contribution_matrix);
    hipMalloc((void**) &alive_cell_count_d, sizeof(int));
    
    

    alive_cell_count_h = 1;

    // printf("%d ",*alive_cell_count_d);

    for(int i=0; i<world_dim*world_dim; i++) {
        world_value_h[i] = 0.0;
        world_id_h[i] = 0;
        world_signal_h[i] = 0;
    }

    for(int i=0; i<alive_cell_count_h; i++){
        int indirizzo = rand() % (world_dim*world_dim);
        alive_cell_h[i] = indirizzo;
        world_value_h[indirizzo] = 1.0;
        world_id_h[indirizzo] = i + 1;
    }

    for(int i=0; i<n_modelli*world_dim; i++){
        contribution_matrix_h[i] = 0;
    }

    for(int i=0; i<n_weights*n_modelli; i++){
        weight_h[i] = random_float(-1,1);
    }

    for(int i=0; i<n_baias*n_modelli; i++){
        baias_h[i] = random_float(-1,1);
    }

    hipMemcpy(world_value_d,           world_value_h,        size_world_float,               hipMemcpyHostToDevice);
    hipMemcpy(world_signal_d,          world_signal_h,       size_world_float,               hipMemcpyHostToDevice);
    hipMemcpy(world_id_d,              world_id_h,           size_world_int,                 hipMemcpyHostToDevice);
    hipMemcpy(alive_cell_d,            alive_cell_h,         size_world_int,                 hipMemcpyHostToDevice);
    hipMemset(inputs_d,                0,                    size_inputs);
    hipMemset(outputs_d,               0,                    size_outputs);
    hipMemcpy(weight_d,                weight_h,             size_weight,                    hipMemcpyHostToDevice);
    hipMemcpy(baias_d,                 baias_h,              size_bias,                      hipMemcpyHostToDevice); 
    hipMemset(contribution_matrix_d,   0,                    size_contribution_matrix);
    hipMemcpy(alive_cell_count_d,      &alive_cell_count_h,  sizeof(int),                    hipMemcpyHostToDevice);

    int Step = 0;
    while(alive_cell_count_h > 0 && Step < n_step){
        Step++;
        printf("STEP NUMERO:            %4d \n",Step);
        // hipDeviceSynchronize();
        int offset_alive_cell = 0;
        while(offset_alive_cell<alive_cell_count_h){

            int max = numero_workspace<alive_cell_count_h-offset_alive_cell?numero_workspace:alive_cell_count_h-offset_alive_cell;

            for(int workspace_idx=0; workspace_idx<max; workspace_idx++){

                int offset_workspace_in = input_dim*workspace_idx;
                int offset_workspace_out = output_dim*workspace_idx;

                launch_vision(
                    world_value_d,
                    world_id_d,
                    world_signal_d,
                    world_dim,
                    alive_cell_d+offset_alive_cell,
                    visione,
                    inputs_d+offset_workspace_in,
                    stream
                );

                launch_NN_forward(
                    inputs_d+offset_workspace_in,
                    outputs_d+offset_workspace_out,
                    weight_d,
                    n_weights,
                    baias_d,
                    n_baias,
                    structure,
                    offset_alive_cell,
                    alive_cell_d,
                    world_id_d,
                    n,
                    stream
                );
                
                
                launch_output_elaboration(
                    world_value_d,
                    world_signal_d,
                    world_id_d,
                    contribution_matrix_d,
                    outputs_d+offset_workspace_out,
                    alive_cell_d,
                    world_dim,
                    n_modelli,
                    output_dim,
                    offset_alive_cell,
                    stream
                );
                

                offset_alive_cell++;

            }
            hipDeviceSynchronize();
            printf("Cellule fino a %d \n",offset_alive_cell);
        }

        
        hipDeviceSynchronize();
        launch_world_update(
            world_value_d,
            world_id_d,
            contribution_matrix_d,
            alive_cell_d,
            world_dim,
            n_modelli,
            alive_cell_count_d,
            stream
        );

        // printf("launch_world_update \n");
        hipDeviceSynchronize();
        
        launch_cellule_cleanup(
            alive_cell_d,
            alive_cell_count_d,
            world_id_d,
            stream
        );
        
        // printf("launch_cellule_cleanup \n");

        hipMemcpy(world_value_h,           world_value_d,          size_world_float,                 hipMemcpyDeviceToHost);
        hipMemcpy(world_signal_h,          world_signal_d,         size_world_float,                 hipMemcpyDeviceToHost);
        hipMemcpy(world_id_h,              world_id_d,             size_world_int,                   hipMemcpyDeviceToHost);
        hipMemcpy(alive_cell_h,            alive_cell_d,           size_world_int,                   hipMemcpyDeviceToHost);
        hipMemcpy(inputs_h,                inputs_d,               size_inputs,                      hipMemcpyDeviceToHost); 
        hipMemcpy(outputs_h,               outputs_d,              size_outputs,                     hipMemcpyDeviceToHost);
        hipMemcpy(weight_h,                weight_d,               size_weight,                      hipMemcpyDeviceToHost);
        hipMemcpy(baias_h,                 baias_d,                size_bias,                        hipMemcpyDeviceToHost); 
        hipMemcpy(contribution_matrix_h,   contribution_matrix_d,  size_contribution_matrix,         hipMemcpyDeviceToHost); 
        hipMemcpy(&alive_cell_count_h,     alive_cell_count_d,     sizeof(int),                      hipMemcpyDeviceToHost);

        
        std::cout << "\n=== WORLD VALUE ===\n";
        for (int y = 0; y < world_dim; y++) {
            for (int x = 0; x < world_dim; x++) {
                printf("%.4f ", world_value_h[y * world_dim + x]);
            }
            std::cout << "\n";
        }

        std::cout << "\n=== WORLD ID ===\n";
        for (int y = 0; y < world_dim; y++) {
            for (int x = 0; x < world_dim; x++) {
                printf("%4d ",world_id_h[y * world_dim + x]);
            }
            std::cout << "\n";
        }
            /*

        std::cout << "\n=== CONTRIBUTION MATRIX===\n";
        for(int i=0; i<world_dim; i++){
            for(int j=0; j<world_dim; j++){
                printf("( ");
                for(int k=0; k<n_modelli; k++){
                    printf("%.2f ",contribution_matrix_h[(i * world_dim) + j + (k * world_dim*world_dim) ]);
                }
                printf(") ");
            }
            printf("\n");
        }
        printf("\n");

        std::cout << "\n=== WORLD SIGNAL ===\n";
        for (int y = 0; y < world_dim; y++) {
            for (int x = 0; x < world_dim; x++) {
                printf("%.4f ",world_signal_h[y * world_dim + x]);
            }
            std::cout << "\n";
        }
            */
        
        /*

        std::cout << "\n=== OUTPUTS (workspace x input_dim) ===\n";
        for (int ws = 0; ws < 1; ws++) {
            std::cout << "Workspace " << ws << ": \n";
            for (int j = 0; j < output_dim; j++) {
                printf("%.4f ",outputs_h[ws * output_dim + j]);
            }
            std::cout << "\n";
        }

        */

        std::cout << "\n=== ALIVE CELLS ===\n";
        for (int i = 0; i < alive_cell_count_h; i++) {
            std::cout << "Alive[" << i << "] = " << alive_cell_h[i] << "\n";
        }

        /* 
        std::cout << "\n=== INPUTS (workspace x input_dim) ===\n";
        for (int ws = 0; ws < numero_workspace; ws++) {
            std::cout << "Workspace " << ws << ": \n";
            for (int j = 0; j < input_dim; j++) {
                printf("%.4f ",inputs_h[ws * input_dim + j]);
                }
                std::cout << "\n";
                }

        std::cout << "\n=== CONTRIBUTION MATRIX===\n";
        for(int i=0; i<world_dim; i++){
            for(int j=0; j<world_dim; j++){
                printf("( ");
                for(int k=0; k<n_modelli; k++){
                    printf("%.4f ",contribution_matrix_h[(i * world_dim) + j + (k * world_dim*world_dim) ]);
                }
                printf(") ");
            }
            printf("\n");
        }
        printf("\n");

        *//*

        printf("\n===Alive Cell===\n");
        printf("%4d \n",alive_cell_count_h);

        
        printf("\n=== MODELLO ===\n");
        printf("BIASES :  %d\n", n_baias);
        printf("WEIGHTS:  %d\n", n_weights);

        printf("\n=== DEVICE POINTERS ===\n");
        printf("WORLD VALUE  (float*): %p\n", (void*)world_value_d);
        printf("WORLD SIGNAL (float*): %p\n", (void*)world_signal_d);
        printf("WORLD ID     (int*)  : %p\n", (void*)world_id_d);
        printf("ALIVE CELL   (int*)  : %p\n", (void*)alive_cell_d);
        printf("INPUTS       (float*): %p\n", (void*)inputs_d);    
        printf("OUTPUTS      (float*): %p\n", (void*)outputs_d);   
        printf("WEIGHTS      (float*): %p\n", (void*)weight_d);   
        printf("BIASES       (float*): %p\n", (void*)baias_d); 
        printf("CONTRIBUTION (float*): %p\n", (void*)contribution_matrix_d);  
        */
        
    }

    return 0;

}