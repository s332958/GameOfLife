#include "hip/hip_runtime.h"
#include "NN_kernel.cuh"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

__device__ float relu(float x) {
    return x > 0.0f ? x : 0.0f;
}
__device__ float sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}
__device__ float fast_sigmoid(float x) {
    return 0.5f * (x / (1.0f + fabsf(x))) + 0.5f;  
}

// ============================================================================

__global__ void vision_kernel(
    float* world_value,
    int* world_id,
    float* world_signaling,
    int dim_world,
    int* cell_idx,
    int dim_window,
    float* workspace,
    int limit_workspace_cell,
    int dim_input
) 
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    int index_cell = index / dim_input;
    int vision_index = index % dim_input;
    
    if (index >= dim_input*limit_workspace_cell) return;

    int dim_window_sq = dim_window*dim_window;
    int radius = dim_window / 2;

    int center_index = cell_idx[index_cell];

    int vision_window_index = vision_index % dim_window_sq;
    int visiontype_index = vision_index / dim_window_sq;

    int center_x = center_index % dim_world;
    int center_y = center_index / dim_world;

    int vision_x = vision_window_index % dim_window;
    int vision_y = vision_window_index / dim_window;

    int real_vision_x = (center_x + (vision_x - radius) + dim_world) % dim_world;
    int real_vision_y = (center_y + (vision_y - radius) + dim_world) % dim_world;

    int real_vision_index = real_vision_y*dim_world + real_vision_x;

    int cell_ID = world_id[center_index];
    int vision_ID = world_id[real_vision_index];
    if(visiontype_index == 0){
        if(cell_ID == vision_ID){            
            workspace[index] = world_signaling[real_vision_index];
        }else{
            workspace[index] = - world_signaling[real_vision_index];
        }

    }else{
        if(vision_ID == -1){
            workspace[index] = - 1.0f;
        }else{
            workspace[index] = world_value[real_vision_index];
        }

    }
}

// ============================================================================


__global__ void NN_forward_weight_kernel(
    float* input,
    float* output, 
    float* weights, 
    int* cells,
    int* world_id, 
    int n_weights, 
    int limit_workspace_cell,
    int layer1_size,
    int layer2_size,
    int offset_weights
){
        // index del thread 
        int tidx = blockIdx.x * blockDim.x + threadIdx.x;

        // se il thread supera la dim dei weights  allora ritorno 
        if (tidx >= layer1_size * layer2_size * limit_workspace_cell){
            return;
        }
        int cell_index = tidx / (layer1_size * layer2_size);
        int weight_index = tidx % (layer1_size * layer2_size);
       
        int world_index = cells[cell_index];
        int ID = world_id[world_index];

        if(ID==0) printf("KERNEL FOREWARD WEIGHTS ID=%d \n",ID);
                
        int true_weight_index = n_weights * (ID - 1) + weight_index + offset_weights; 
         
        int input_neuron_idx  = weight_index % layer1_size;
        int output_neuron_idx = weight_index / layer1_size;

        int input_index  = cell_index * layer1_size  + input_neuron_idx;
        int output_index = cell_index * layer2_size  + output_neuron_idx;
            
        float weighted = weights[true_weight_index] * input[input_index];

        atomicAdd(&output[output_index], weighted);
        
}

__global__ void NN_forward_bias_kernel(
    float* output, 
    float* biases, 
    int* cells,
    int* world_id, 
    int n_biases, 
    int limit_workspace_cell,
    int layer2_size,
    int offset_biases
){
        // index del thread 
        int tidx = blockIdx.x * blockDim.x + threadIdx.x;
       
        if (tidx >= layer2_size * limit_workspace_cell){
            return;
        }
        int cell_index = tidx / layer2_size;
        int bias_index = tidx % layer2_size;

        int world_index = cells[cell_index];
        int ID = world_id[world_index];

        int true_bias_index = n_biases * (ID - 1) + bias_index + offset_biases;

        output[tidx] += biases[true_bias_index];
        if(ID==0) printf("KERNEL FOREWARD BIAS ID=%d \n",ID);

        // applico la relu ad ogni cella di output modificata 
        output[tidx] = fast_sigmoid(output[tidx]);
        
}

//===================================================================================


__global__ void output_elaboration_kernel(
    float* world_value,
    float* world_signal,
    int* world_id,
    float* contribution_matrix,
    float* outputs,   
    int* cells,
    int dim_world, 
    int number_of_creatures,
    int output_size,
    int dim_window,
    int limit_workspace_cell,
    float energy_fraction
){  
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    int index_cell = index / output_size;
    int output_index = index % output_size;
    
    if (index >= output_size*limit_workspace_cell) return;
    
    int center_index = cells[index_cell];

    float output = outputs[index];
    output = sigmoid(output);

    if (output_index == (output_size - 1)){
        world_signal[center_index] = output;
        return;
    }

    float center_value = world_value[center_index];

    int dim_window_sq = dim_window*dim_window;
    float final_output = center_value * energy_fraction / (dim_window_sq - 1) * output;
    int radius = dim_window / 2;

    int vision_window_index = output_index % dim_window_sq;

    int center_x = center_index % dim_world;
    int center_y = center_index / dim_world;

    int vision_x = vision_window_index % dim_window;
    int vision_y = vision_window_index / dim_window;

    int real_vision_x = (center_x + (vision_x - radius) + dim_world) % dim_world;
    int real_vision_y = (center_y + (vision_y - radius) + dim_world) % dim_world;

    int cell_ID = world_id[center_index];
    int ID_offset = (cell_ID - 1) * (dim_world * dim_world);

    int real_vision_index = real_vision_y*dim_world + real_vision_x + ID_offset;

    atomicAdd(&contribution_matrix[real_vision_index], final_output);
    atomicAdd(&world_value[center_index], - final_output); 
}

// ===================================================================================

__global__ void compute_energy_and_occupation_kernel(
    float* world_value,
    int* world_id,
    float* occupation_vector,
    float* energy_vector,
    int world_dim,
    int n_creature
) {

    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index >= world_dim * world_dim) return;

    int id = world_id[index] - 1;

    if (id < 0)return;
    
    atomicAdd(&occupation_vector[id], 1.0f/float(world_dim));
    atomicAdd(&energy_vector[id], world_value[index]/float(world_dim));

}

// ==================================================================================

// PREMESSA:
// la struttura dei blocchi è importante, il numero di thread dentro ad un blocco, indica il numero di pesi/biases che ogni blocco genetico possiede
__global__ void recombine_models_kernel(
    float *weights, float *biases,
    float *new_weights, float *new_biases,
    int num_weights_per_model, int num_bias_per_model,
    int model1_idx, int model2_idx, int output_idx,
    float mutation_prob,
    float mutation_range,
    unsigned long seed)
{
    __shared__ int gen_id;

    // Calcolo indice thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_genes = num_weights_per_model + num_bias_per_model;
    // faccio una return dei thread che superano la somma di pesi+biases
    if (idx >= total_genes) return;

    // creazione d curandstate per generare valori casuali su device
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // il primo thread di ogni blocco si occupa di generare il numero del genitore del blocco genetico
    if(threadIdx.x==0){
        int gen = hiprand(&state) % 2;
        gen_id = gen==0?model1_idx:model2_idx;
    }

    __syncthreads();

    int idx_model_param_gen = -1;

    // caso in cui stiamo analizzando i pesi (indice < numero pesi modello)
    if (idx < num_weights_per_model) {

        // calcolo l'indice dei geni (pesi singoli) dal blocco genetico del genitore
        idx_model_param_gen = (gen_id * num_weights_per_model) + idx;

        // carico il valore del gene 
        float gene_value = weights[idx_model_param_gen];

        // genero un numero casuale per definire se il gene subisce una mutazione (caso per cui viene superata la soglia di mutazione)
        if (hiprand_uniform(&state) < mutation_prob) {
            // calcolo il delta della variazione che va da -mutation_range a mutation_range
            float delta = (hiprand_uniform(&state) * 2.0f - 1.0f) * mutation_range;
            // applico il delta
            gene_value += delta;
        }

        // trovo l'indice per scrivere il nuovo valore sul nuovo modello e lo aggiorno
        int idx_model_param_out = (output_idx * num_weights_per_model + idx);
        new_weights[idx_model_param_out] = gene_value;

    }else{

        //caso in cui siamo nei bias (indice > numero peso modelli) 
        //calcolo indice gene genitore come per i pesi ma si toglie l'offset del numero di pesi (questo perche i thread dei biases sono tutti dopo i pesi)
        idx_model_param_gen = (gen_id * num_bias_per_model) + idx - num_weights_per_model;

        // carico il valore del gene in un registro
        float gene_value = biases[idx_model_param_gen];

        // genero un valore casuale che se supera la soglia allora indica la mutazione del gene
        if (hiprand_uniform(&state) < mutation_prob) {
            // calcolo il valore di mutzione del gene come fatto in precedenza 
            float delta = (hiprand_uniform(&state) * 2.0f - 1.0f) * mutation_range;
            // aggiorno il valore del nuovo gene
            gene_value += delta;
        }

        // trovo l'indice su dove va scritto il bias appena calcolato e lo aggiorno
        int idx_model_param_out = (output_idx * num_bias_per_model) + idx - num_weights_per_model;
        new_biases[idx_model_param_out] = gene_value;

    }

}

//=================================================================================

// Wrapper kernel visione
void launch_vision(                 
    float* world_value,             
    int* world_id,                  
    float* world_signaling,        
    int dim_world,                 
    int* cell_idx,                 
    int dim_window,                       
    float* input_workspace,               
    int limit_workspace_cell,
    hipStream_t stream
){

    int n_thread_per_block = 1024;
    int dim_input = dim_window * dim_window * 2;
    int thread_number = dim_input * limit_workspace_cell;
    int n_block = (thread_number + n_thread_per_block - 1) / n_thread_per_block;
    vision_kernel<<<n_block,n_thread_per_block,0,stream>>>(
        world_value,
        world_id,
        world_signaling,
        dim_world,
        cell_idx,
        dim_window,
        input_workspace,
        limit_workspace_cell,
        dim_input
    );
    //if(hipGetLastError()!=hipError_t::hipSuccess) printf("errori vision_kernel: %s\n",hipGetErrorString(hipGetLastError()));          


}

// ===================================================================================================

// Wrapper kernel NN_forward
void launch_NN_forward(                           
    float* input_workspace,                  
    float* output_workspace,                   
    int workspace_size,
    float* weights,                               
    int n_weights,                                
    float* biases,                                 
    int n_biases,                                   
    int* structure,     
    int limit_workspace_cell,
    int *cells,                                     
    int *world_id,                                 
    int dim_structure,                              
    hipStream_t stream    
){
    int n_thread_per_block = 1024;
    int layer1_size = 0;
    int layer2_size = 0;
    int weight_offset = 0;
    int biases_offset = 0;

    for(int i=0; i < (dim_structure-1); i++){

        layer1_size = structure[i];
        layer2_size = structure[i + 1];

        int thread_number = layer1_size * layer2_size * limit_workspace_cell;             

        int n_block = (thread_number + n_thread_per_block - 1) / n_thread_per_block;

        NN_forward_weight_kernel<<<n_block, n_thread_per_block, 0 , stream>>>(
            input_workspace,
            output_workspace, 
            weights, 
            cells, 
            world_id, 
            n_weights, 
            limit_workspace_cell,
            layer1_size, 
            layer2_size, 
            weight_offset
        );  

        thread_number = layer2_size * limit_workspace_cell;             
        n_block = (thread_number + n_thread_per_block - 1) / n_thread_per_block;


        NN_forward_bias_kernel<<<n_block, n_thread_per_block, 0 , stream>>>(
            output_workspace, 
            biases, 
            cells, 
            world_id, 
            n_biases, 
            limit_workspace_cell,
            layer2_size, 
            biases_offset
        ); 
        //if(hipGetLastError()!=hipError_t::hipSuccess) printf("errori NN_forward_kernel: %s\n",hipGetErrorString(hipGetLastError()));
        hipMemcpy(input_workspace, output_workspace, workspace_size*limit_workspace_cell, hipMemcpyDeviceToDevice);
        
        if (i < (dim_structure-2)){
            hipMemset(output_workspace, 0, workspace_size*limit_workspace_cell);
        }
        weight_offset += layer1_size*layer2_size;
        biases_offset += layer2_size;

    }

}

// ===================================================================================================

//Wrapper kernel output_elaboration
void launch_output_elaboration(              
    float* world_value,                      
    float* world_signal,                     
    int* world_id,                        
    float* contribution_matrix,           
    float* output_workspace,              
    int* cells,                           
    int world_dim,                        
    int number_of_creatures,              
    int output_size,   
    int limit_workspace_cell,
    float energy_fraction,
    hipStream_t stream
){
    int window_size = sqrt(output_size - 1);
    int n_thread_per_block = 1024;
    int thread_number = limit_workspace_cell * output_size;
    int n_block = (thread_number + n_thread_per_block - 1) / n_thread_per_block;

    output_elaboration_kernel<<<n_block, n_thread_per_block, 0, stream>>>(
        world_value,
        world_signal,
        world_id,
        contribution_matrix,
        output_workspace,
        cells,
        world_dim,
        number_of_creatures,
        output_size,
        window_size,
        limit_workspace_cell,
        energy_fraction
    );
    //if(hipGetLastError()!=hipError_t::hipSuccess) printf("errori output_elaboration_kernel: %s\n",hipGetErrorString(hipGetLastError()));
    

}

// ===================================================================================================

// Wrapper compute energy and occupation for evaluation
void launch_compute_energy_and_occupation(
    float* world_value,
    int* world_id,
    float* occupation_vector,
    float* energy_vector,
    int world_dim,
    int n_creature,
    hipStream_t stream
){

    int n_thread_per_block = 1024; 
    int thread_number = world_dim * world_dim;

    int n_block = (thread_number + n_thread_per_block - 1) / n_thread_per_block;

    compute_energy_and_occupation_kernel<<<n_block,n_thread_per_block,0,stream>>>(
        world_value,
        world_id,
        occupation_vector,
        energy_vector,
        world_dim,
        n_creature
    );


}

// ==================================================================================================

// Wrapper: recombine_model
void launch_recombine_models_kernel(
    float *d_weights, float *d_biases,
    float *d_new_weights, float *d_new_biases,
    int num_weights_per_model, int num_bias_per_model,
    int model1_idx, int model2_idx, int output_idx,
    float gen_x_block,
    float mutation_prob,
    float mutation_range,
    unsigned long seed,
    hipStream_t stream) 
{
    // Numero totale di geni (pesi + bias)
    int total_genes = num_weights_per_model + num_bias_per_model;

    // Imposta configurazione kernel
    int threads_per_block = gen_x_block*total_genes +1;
    if(threads_per_block>1024) threads_per_block = 1024;
    int num_blocks = (total_genes + threads_per_block - 1) / threads_per_block;

    // Lancia il kernel
    recombine_models_kernel<<<num_blocks, threads_per_block, 0, stream>>>(
        d_weights,
        d_biases,
        d_new_weights,
        d_new_biases,
        num_weights_per_model,
        num_bias_per_model,
        model1_idx,
        model2_idx,
        output_idx,
        mutation_prob,
        mutation_range,
        seed
    );

}














// ===============================================================

// ================================================================================================

__global__ void generate_clone_creature_kernel(
    float *weight_starting_model,
    float *biases_starting_model,
    float *weights_vector,
    float *biases_vector,
    float *varation_weights_vector,
    float *varation_biases_vector,
    int    n_weights,
    int    n_biases,
    int    n_creature,
    float  std,
    hiprandState_t *states
){

    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if(idx >= n_creature*n_weights) return;

    hiprandState state = states[threadIdx.x];

    float varation = (hiprand_uniform(&state) * 2) -1;
    varation = varation * std; 

    int id_creature = idx / n_weights;
    int param_original_idx = idx % n_weights;
    int final_pos = id_creature*n_weights + param_original_idx;

    varation_weights_vector[final_pos] = varation;
    weights_vector[final_pos] = weight_starting_model[param_original_idx] + varation;


    if(idx >= n_creature*n_biases) return;

    varation = (hiprand_uniform(&state) * 2) -1; 

    id_creature = idx / n_biases;
    param_original_idx = idx % n_biases;
    final_pos = id_creature*n_biases + param_original_idx;

    varation_biases_vector[final_pos] = varation;
    biases_vector[final_pos] = biases_starting_model[param_original_idx] + varation;


}


void launch_generate_clone_creature(
    float *weight_starting_model,
    float *biases_starting_model,
    float *weights_vector,
    float *biases_vector,
    float *varation_weights_vector,
    float *varation_biases_vector,
    int    n_weights,
    int    n_biases,
    int    n_creature,
    float  std,
    hipStream_t stream,
    hiprandState_t *states
){

    int n_thread = n_weights*n_creature;
    if(n_thread>1024) n_thread = 1024;
    int n_block = (n_weights + n_thread -1) / n_thread;

    generate_clone_creature_kernel<<<n_block,n_thread,0,stream>>>(
        weight_starting_model,
        biases_starting_model,
        weights_vector,
        biases_vector,
        varation_weights_vector,
        varation_biases_vector,
        n_weights,
        n_biases,
        n_creature,
        std,
        states
    );

}



// ========================================================================================================


__global__ void update_model_kernel(
    float *weight_starting_model,
    float *biases_starting_model,
    float *varation_weights_vector,
    float *varation_biases_vector,
    float *score_vector,
    int    n_weights,
    int    n_biases,
    int    n_creature,
    float  alpha,
    float  std,
    int n_steps
){

    __shared__ float shared_mem;

    int creature_idx = threadIdx.x;
    int params_idx = blockIdx.x;

    if(creature_idx >= n_creature || params_idx >= n_biases+n_weights) return;

    if(threadIdx.x==0){
        shared_mem = 0;
    }

    __syncthreads();

    if(blockIdx.x < n_weights){

        int val = varation_weights_vector[params_idx] * score_vector[creature_idx];
        atomicAdd(&shared_mem,val);

        __syncthreads();

        val = shared_mem;

        val = (val * alpha) / (n_creature * std);
        weight_starting_model[params_idx] += val;

    }else{

        params_idx -= n_weights;

        int val = varation_biases_vector[params_idx] * score_vector[creature_idx];
        atomicAdd(&shared_mem,val);

        __syncthreads();

        val = shared_mem;

        val = (val * alpha) / (n_creature * std * n_steps);
        biases_starting_model[params_idx] += val;

    }

}


void launch_update_model(
    float *weight_starting_model,
    float *biases_starting_model,
    float *varation_weights_vector,
    float *varation_biases_vector,
    float *score_vector,
    int    n_weights,
    int    n_biases,
    int    n_creature,
    float  alpha,
    float  std,
    int n_steps,
    hipStream_t stream
){

    int n_thread = n_creature;
    if(n_thread>1024) n_thread = 1024;
    int n_block = n_weights+n_biases;

    update_model_kernel<<<n_block,n_thread,0,stream>>>(
        weight_starting_model,
        biases_starting_model,
        varation_weights_vector,
        varation_biases_vector,
        score_vector,
        n_weights,
        n_biases,
        n_creature,
        alpha,
        std,
        n_steps
    );

}