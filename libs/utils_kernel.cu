#include "hip/hip_runtime.h"
#include "utils_kernel.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <iostream>



// ============================================================================

__global__ void fill_random_kernel(float* d_vec, int start, int finish, float minVal, float maxVal, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= finish - start) return;

    int real_idx = idx + start;

    // Setup generator per thread
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    float rand_uniform = hiprand_uniform(&state); // [0,1)

    d_vec[real_idx] = minVal + rand_uniform * (maxVal - minVal);
}



// Wrapper: fill vettore random con calcolo griglia/thread
void launch_fill_random_kernel(float* d_vec, int start, int finish, float minVal, float maxVal,
                                unsigned long seed, hipStream_t stream) {

    int threads = 1024;
    int n = finish - start;
    int blocks = (n + threads - 1) / threads;

    fill_random_kernel<<<blocks, threads, 0, stream>>>(d_vec, start, finish, minVal, maxVal, seed);

}
template <typename T>
__global__ void resetKernel(T* d_vec, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    d_vec[idx] = T(0);
}

template <typename T>
void launch_reset_kernel(T* d_vec, int n, hipStream_t stream) {
    int threads = 1024;
    int blocks = (n + threads - 1) / threads;
    resetKernel<T><<<blocks, threads, 0, stream>>>(d_vec, n);
}

// ========== Esplicit template instantiation ==========
template void launch_reset_kernel<float>(float*, int, hipStream_t);
template void launch_reset_kernel<int>(int*, int, hipStream_t);
