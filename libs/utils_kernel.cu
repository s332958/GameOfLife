#include "hip/hip_runtime.h"
#include "utils_kernel.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <iostream>



// ============================================================================

// funtion for generating random number in a vector in GPU
__global__ void fill_random_kernel(float* d_vec, int start, int finish, float minVal, float maxVal, hiprandState states[]) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= finish - start) return;

    int real_idx = idx + start;

    // Setup generator per thread
    hiprandState state = states[threadIdx.x];

    float rand_uniform = hiprand_uniform(&state); // [0,1)

    d_vec[real_idx] = minVal + rand_uniform * (maxVal - minVal);
}



// Wrapper: fill vettore random con calcolo griglia/thread
void launch_fill_random_kernel(float* d_vec, int start, int finish, float minVal, float maxVal,
                                hiprandState states[],
                                hipStream_t stream) {

    int threads = 1024;
    int n = finish - start;
    int blocks = (n + threads - 1) / threads;

    fill_random_kernel<<<blocks, threads, 0, stream>>>(d_vec, start, finish, minVal, maxVal, states);

}
template <typename T>
__global__ void resetKernel(T* d_vec, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    d_vec[idx] = T(0);
}

template <typename T>
void launch_reset_kernel(T* d_vec, int n, hipStream_t stream) {
    int threads = 1024;
    int blocks = (n + threads - 1) / threads;
    resetKernel<T><<<blocks, threads, 0, stream>>>(d_vec, n);
}

// ========== Esplicit template instantiation ==========
template void launch_reset_kernel<float>(float*, int, hipStream_t);
template void launch_reset_kernel<int>(int*, int, hipStream_t);































































// NUOVI TEST





__global__ void init_curandstates_kernel(hiprandState* states, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, threadIdx.x - blockIdx.x, &states[idx]);
}

void launch_init_curandstates(hiprandState d_states[], int total_threads, unsigned long seed, hipStream_t stream) {
    int threads_per_block = 256;
    int blocks = (total_threads + threads_per_block - 1) / threads_per_block;

    init_curandstates_kernel<<<blocks, threads_per_block, 0, stream>>>(d_states, seed);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error in init_curandstates_kernel: %s\n", hipGetErrorString(err));
    }
}
