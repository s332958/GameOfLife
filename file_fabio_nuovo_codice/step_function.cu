#include "kernel_mondo.cu"
#include "kernel_neuralNet.cu"

#include <hip/hip_runtime.h>
#include <stdio.h>


//================================================================================================


void check_error(char *descrizione){
    hipError_t errore = hipGetLastError();
    if(errore!=hipSuccess){
        printf("Errore cuda in '%s': %s\n",descrizione,hipGetErrorString(errore));
    }
}



// ================================================================================================

void load_world_on_GPU(
    float *world_val_h, int *world_id_h,
    float **world_val_d, int **world_id_d,
    int dim, hipStream_t stream, int cc
                    ) {

    int totdim = dim * dim;

    // Alloco memoria sulla GPU
    if(cc >= 7) hipMallocAsync((void**)world_val_d, totdim * sizeof(float), stream);
    else hipMalloc((void**)world_val_d, totdim * sizeof(float));
    check_error("Allocazione mondo valori");

    if(cc >= 7) hipMallocAsync((void**)world_id_d, totdim * sizeof(int), stream);
    else hipMalloc((void**)world_id_d, totdim * sizeof(int));
    check_error("Allocazione mondo id");

    // Copio dati da host a device
    hipMemcpyAsync(*world_val_d, world_val_h, totdim * sizeof(float), hipMemcpyHostToDevice, stream);
    check_error("Caricamento mondo valori");
    
    hipMemcpyAsync(*world_id_d, world_id_h, totdim * sizeof(int), hipMemcpyHostToDevice, stream);
    check_error("Caricamento mondo valori");
}




void setup_world(float *mondo_val_d, int *mondo_id_d, int dim_mondo){
    
    // funzione generazione ostacoli
    // funzione generazione del cibo

}

// ===========================================================================================

void add_creatures(
    float *world_value_d, int world_id_d, int dim_world, 
    int n_creature, 
    int cc, hipStream_t stream
                ){

    // kernel che in base al numero di creature va ad instanziare 

}

// ==================================================================================================

void load_behaviour_on_GPU(
    Creature *creatures, int n_creature, float **weight_d, float **bias_d,
    int cc, hipStream_t stream
){

    int totdimw = creatures[0].dim_weight * n_creature;
    int totdimb = creatures[0].dim_bias * n_creature;

    if(cc >= 7) hipMallocAsync((void**)&weight_d, totdimw * sizeof(float), stream);
    else hipMalloc((void**)&weight_d, totdimw * sizeof(float));
    check_error("Allocazione pesi di tutte le creature creature");

    if(cc >= 7) hipMallocAsync((void**)&bias_d, totdimb * sizeof(float), stream);
    else hipMalloc((void**)&bias_d, totdimb * sizeof(float));
    check_error("Allocazione bias di tutte le creature creature");

    for(int i=0; i<n_creature; i++){
        Creature creature = creatures[i];

        hipMemcpyAsync(*weight_d, creature.weight_model, creature.dim_weight * sizeof(float), hipMemcpyHostToDevice, stream);
        check_error("Caricamento mondo valori");
        
        hipMemcpyAsync(*bias_d, creature.bias_model, creature.dim_bias * sizeof(int), hipMemcpyHostToDevice, stream);
        check_error("Caricamento mondo valori");
    }

}